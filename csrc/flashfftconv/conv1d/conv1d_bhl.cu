#include "hip/hip_runtime.h"
// Copyright (c) 2023 Dan Fu, Hermann Kumbong

// Simple 1D depthwise convolution implementation with dilation and stride = 1
#include "shared.h"

const uint BX = 256;
const uint BY = 1;
const uint BZ = 1;

const uint TILE_SIZE_L = 4;
const uint TILE_SIZE_D = 1;

template<typename T, typename U>
__forceinline__ __device__ T _conv1d_k_3(const T* u, const U* weights, const U* bias, uint padding, uint l, uint d, uint L, uint D, uint K)
{
    T tmp;
    T weight;

    set_value(&tmp, bias[d]);

    int idx = l - padding;

    if(idx >= 0 && idx < L){
        set_value(&weight, weights[0]);
        tmp = __hfma(u[d * L + idx], weight, tmp);
    }
    
    idx++;
    if(idx >= 0 && idx < L){
        set_value(&weight, weights[1]);
        tmp = __hfma(u[d * L + idx], weight, tmp);
    }

    idx++;
    if(idx >= 0 && idx < L){
        set_value(&weight, weights[2]);
        tmp = __hfma(u[d * L + idx], weight, tmp);
    }

    return tmp;
}

template<typename T, typename U>
__global__ void conv1d_kernel(
    const T *__restrict__ u,
    const U *__restrict__ weights,
    const U *__restrict__ bias,
    T *__restrict__ out,
    uint padding,
    uint B,
    uint L,
    uint D,
    uint K,
    uint L_out
    )
{
    const int b = blockIdx.z * blockDim.z + threadIdx.z;
    const int d = blockIdx.y * blockDim.y * TILE_SIZE_D + threadIdx.y;
    const int l_offset = blockIdx.x * blockDim.x * TILE_SIZE_L + threadIdx.x;
    
    T tmp; 
    T weight;

    int idx;
    int l;

    for(int l_tile = 0; l_tile < TILE_SIZE_L; l_tile++){
        l = l_offset + l_tile * blockDim.x;

        set_value(&tmp, bias[d]);

        if(d < D && l < L_out && b < B){
            if(K == 3){
                out[b * L_out * D + d * L_out + l] = _conv1d_k_3(u + b * L * D, weights + d * K, bias, padding, l, d, L, D, K);
            } else{
                for(int k = 0; k < K; k++){
                    idx = l - padding + k;
                    if(idx >= 0 && idx < L){
                        set_value(&weight, weights[d * K + k]);
                        tmp = __hfma(u[b * L_out * D + d * L + idx], weight, tmp);
                    }
                }
                out[b * L_out * D + d * L_out + l] = tmp;
            
            }
        }
    }
    
}

torch::Tensor conv1d_cuda_bhl(
    torch::Tensor u,
    torch::Tensor weight,
    torch::Tensor bias,
    uint padding)
{
    const uint b = u.size(0);
    const uint d = u.size(1);
    const uint l = u.size(2);


    const uint k = weight.size(1);

    uint l_out = (l + 2 * padding - k + 1);
    
    dim3 blockDims(BX, BY, BZ);

    dim3 gridDims(ceil(l_out * 1.0 / (BX * TILE_SIZE_L) ), ceil((d * 1.0) / (BY * TILE_SIZE_D)), ceil((b * 1.0) / BZ));

    torch::Tensor out = torch::empty({b, d, l_out}, u.options());

    DISPATCH_FLOAT_AND_HALF_AND_BF16(u.scalar_type(), weight.scalar_type(),
        "depthwise conv 1d fwd bhl",
        ([&]
            { conv1d_kernel<input_t, weight_t><<<gridDims, blockDims>>>(
                    static_cast<input_t *>(u.data_ptr()),
                    static_cast<weight_t *>(weight.data_ptr()),
                    static_cast<weight_t *>(bias.data_ptr()),
                    static_cast<input_t *>(out.data_ptr()),
                    padding,
                    b,
                    l,
                    d,
                    k,
                    l_out
                    ); 
            }
        )
    );

    return out;
}