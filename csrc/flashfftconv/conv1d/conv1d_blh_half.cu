#include "hip/hip_runtime.h"
// Copyright (c) 2023 Dan Fu, Hermann Kumbong

// Simple 1D depthwise convolution implementation with dilation and stride = 1

#include <torch/extension.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <hip/hip_fp16.h>

const uint STRIDE = 1;
const uint DILATION = 1;

//For max perf, tune for your GPU and batch size, and datatype etc
const uint BX = 512;
const uint BY = 1;
const uint BZ = 1;

const uint TILE_SIZE_Y = 4;
const uint TILE_SIZE_X = 2;

// Trick to do padding in place without actually creating a new tensor
__forceinline__ __device__ __half2 get_u(const __half2 *__restrict__ u, uint L_eff, uint l, uint p, uint b, uint k, uint d, uint L, uint D, uint K)
{
    return l + k < p || l + k > L_eff - (p + 1) ? __float2half2_rn(0.0f) : u[b * L * D + (l + k - p) * D + d];
}

//manually unrolling loop for k = 3 leads to good perf, can easily extend for other values of k if need be
__forceinline__ __device__ __half2 _conv1d_k_3(const __half2* u, const __half2* weights, const __half2* bias, __half2* out, uint padding, uint b, uint l, uint d, uint t, uint L, uint D, uint K, uint L_eff, uint L_out)
{

    __half2 temp_sum = bias[d];

    temp_sum = __hfma2(get_u(u, L_eff, l + t, padding, b, 0, d, L, D, K), weights[0 * D + d], temp_sum);
    temp_sum = __hfma2(get_u(u, L_eff, l + t, padding, b, 1, d, L, D, K), weights[1 * D + d], temp_sum);
    out[b * D * L_out  + (l + t) * D + d] = __hfma2(get_u(u, L_eff, l + t, padding, b, 2, d, L, D, K), weights[2 * D + d], temp_sum);

}

__global__ void conv1d_kernel_k_3(
    const __half2 *__restrict__ u,
    const __half2 *__restrict__ weights,
    const __half2 *__restrict__ bias,
    __half2 *__restrict__ out,
    uint padding,
    uint B,
    uint L,
    uint L_out,
    uint L_eff,
    uint D,
    uint K)
{
    const int d_block = blockIdx.x * blockDim.x * TILE_SIZE_X;
    const int l = blockIdx.y * blockDim.y * TILE_SIZE_Y + threadIdx.y * TILE_SIZE_Y;
    const int b = blockIdx.z * blockDim.z + threadIdx.z;

    int d;

    #pragma unroll
        for (int i = 0; i < TILE_SIZE_X; i++)
        {   
            d = d_block + threadIdx.x + i * BX;

            if (d < D && b < B){
                #pragma unroll
                for (int t = 0; t < TILE_SIZE_Y; t++){
                    if (l + t < L_eff - K + 1)
                    {
                        if(K == 3){
                            _conv1d_k_3(u, weights, bias, out, padding, b, l, d, t, L, D, K, L_eff, L_out);
                        }
                    }
                }
            }
        }
}


__global__ void conv1d_kernel(
    const __half2 *__restrict__ u,
    const __half2 *__restrict__ weights,
    const __half2 *__restrict__ bias,
    __half2 *__restrict__ out,
    uint padding,
    uint B,
    uint L,
    uint L_out,
    uint L_eff,
    uint D,
    uint K)
{
    const int d_block = blockIdx.x * blockDim.x * TILE_SIZE_X;
    const int l = blockIdx.y * blockDim.y * TILE_SIZE_Y + threadIdx.y * TILE_SIZE_Y;
    const int b = blockIdx.z * blockDim.z + threadIdx.z;

    int d;

    #pragma unroll
        for (int i = 0; i < TILE_SIZE_X; i++)
        {   
            d = d_block + threadIdx.x + i * BX;

            if (d < D && b < B){
                #pragma unroll
                for (int t = 0; t < TILE_SIZE_Y; t++){
                    if (l + t < L_eff - K + 1)
                    {
                        __half2 temp_sum = bias[d];
                        for(int k = 0; k < K; k++){
                            temp_sum = __hfma2(get_u(u, L_eff, l + t, padding, b, k, d, L, D, K), weights[k * D + d], temp_sum);
                        }
                            out[b * D * L_out  + (l + t) * D + d] = temp_sum;
                    }
                }
            }
        }
}

torch::Tensor conv1d_cuda_blh_half(
    torch::Tensor u,
    torch::Tensor weight,
    torch::Tensor bias,
    uint padding)
{
    const uint b = u.size(0);
    const uint l = u.size(1);
    const uint d = u.size(2);

    const uint k = weight.size(0);

    uint l_eff = l + 2 * padding;

    

    dim3 blockDims(BX, BY, BZ);

    dim3 gridDims(ceil(d * 1.0 / (BX * TILE_SIZE_X * 2) ), ceil((l_eff - k + 1) * 1.0 / (BY * TILE_SIZE_Y)), ceil(b * 1.0 / BZ));


    uint l_out = (l + 2 * padding - k + 1);

    torch::Tensor out = torch::empty({b, l_out, d}, u.options());

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv1d_kernel), hipFuncCachePreferL1);

    if(k==3){
        conv1d_kernel_k_3<<<gridDims, blockDims>>>(
            static_cast<__half2 *>(u.data_ptr()),
            static_cast<__half2 *>(weight.data_ptr()),
            static_cast<__half2 *>(bias.data_ptr()),
            static_cast<__half2 *>(out.data_ptr()),
            padding,
            b,
            l,
            l_out,
            l_eff,
            ceil(d/2),
            k);
    }else{
        conv1d_kernel<<<gridDims, blockDims>>>(
            static_cast<__half2 *>(u.data_ptr()),
            static_cast<__half2 *>(weight.data_ptr()),
            static_cast<__half2 *>(bias.data_ptr()),
            static_cast<__half2 *>(out.data_ptr()),
            padding,
            b,
            l,
            l_out,
            l_eff,
            ceil(d/2),
            k);
    }

    return out;
}