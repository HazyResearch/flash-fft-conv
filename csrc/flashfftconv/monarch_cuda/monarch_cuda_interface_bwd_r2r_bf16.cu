// Copyright (c) 2023 Dan Fu, Hermann Kumbong

#include <torch/extension.h>

#include <vector>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include "kernels_bf16/monarch_cuda_bwd_kernel_r2r_bf16.h"
#include "kernels_fp16/monarch_cuda_shared.h"
#include "kernels_bf16/monarch_cuda_shared_bf16.h"
using namespace nvcuda;

// *************** FOR ERROR CHECKING *******************
#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL( call )                                                                                           \
    {                                                                                                                  \
        auto status = static_cast<hipError_t>( call );                                                                \
        if ( status != hipSuccess )                                                                                   \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                     "with "                                                                                           \
                     "%s (%d).\n",                                                                                     \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     hipGetErrorString( status ),                                                                     \
                     status );                                                                                         \
    }
#endif  // CUDA_RT_CALL
// *************** FOR ERROR CHECKING *******************

#ifndef CUDA_CHECK_ERROR
// Define some error checking macros.
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}
#endif  // CUDA_CHECK_ERROR

#ifndef CHECK_LAST_CUDA_ERROR
#define CHECK_LAST_CUDA_ERROR() checkLastBF16BwdR2R(__FILE__, __LINE__)
void checkLastBF16BwdR2R(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}
#endif  // CHECK_LAST_CUDA_ERROR

std::vector<torch::Tensor>
monarch_conv_bwd_cuda_r2r_bf16_all(
  torch::Tensor dout,
  torch::Tensor x,
  torch::Tensor k_f,
  torch::Tensor f_sqrt_N_fft,
  torch::Tensor twiddle_factors_fft,
  torch::Tensor twid_r2r,
  torch::Tensor f_sqrt_N_ifft,
  torch::Tensor twiddle_factors_ifft,
  c10::optional<torch::Tensor> in_gate,
  c10::optional<torch::Tensor> out_gate,
  uint fftsize,
  uint N,
  uint sqrt_N
){

  uint B = x.size(0);
  uint H = x.size(1);
  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // printf("gridDim.x = %d, gridDim.y = %d\n", gridDim.x, gridDim.y);
  torch::Tensor dx_out = torch::empty({B, H, N}, x.options());
  torch::Tensor dk_f_out;
  torch::Tensor din_gate;
  torch::Tensor dout_gate;

  if(in_gate.has_value()){
    din_gate = torch::empty_like(in_gate.value());
  }

  if(out_gate.has_value()){
    dout_gate = torch::empty_like(out_gate.value());
  }

  switch (fftsize) {
    case 256:
      // if (true) {
      if (B >= 2 && (B % 2) == 0 && (H % 4) == 0) {
        gridDim.x = B / 2;
        gridDim.y = H / 4;
        // gridDim.x = B;
        // gridDim.y = H;

        blockDim.x = 32;
        blockDim.y = 1;

        dk_f_out = torch::empty({B / 2, H, fftsize + 1, 2}, x.options());

        monarch_conv_bwd_cuda_kernel<32, 1, 256, 1, false, 2, 4><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      }
      else if ((H % 4) == 0) {
        gridDim.x = B;
        gridDim.y = H / 4;

        blockDim.x = 32;
        blockDim.y = 1;

        dk_f_out = torch::empty({B, H, fftsize + 1, 2}, x.options());

        monarch_conv_bwd_cuda_kernel<32, 1, 256, 1, false, 1, 4><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      } else {
        gridDim.x = B;
        gridDim.y = H;

        blockDim.x = 32;
        blockDim.y = 1;

        dk_f_out = torch::empty({B, H, fftsize + 1, 2}, x.options());

        monarch_conv_bwd_cuda_kernel<32, 1, 256, 1, false, 1, 1><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      }
      break;
    case 1024:
      // if (true) {
      if (B >= 8 && (B % 8) == 0 && (H % 8) == 0) {
        gridDim.x = B / 8;
        gridDim.y = H / 8;
        // gridDim.x = B;
        // gridDim.y = H;

        dk_f_out = torch::empty({B / 8, H, fftsize + 1, 2}, x.options());

        blockDim.x = 32;
        blockDim.y = 1;
        monarch_conv_bwd_cuda_kernel<32, 1, 1024, 2, false, 8, 8><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      }
      else if (B >= 4 && (B % 4) == 0 && (H % 8) == 0) {
        gridDim.x = B / 4;
        gridDim.y = H / 8;

        dk_f_out = torch::empty({B / 4, H, fftsize + 1, 2}, x.options());

        blockDim.x = 32;
        blockDim.y = 1;
        monarch_conv_bwd_cuda_kernel<32, 1, 1024, 2, false, 4, 8><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      } else if ((H % 8) == 0) {
        gridDim.x = B;
        gridDim.y = H / 8;

        dk_f_out = torch::empty({B, H, fftsize + 1, 2}, x.options());

        blockDim.x = 32;
        blockDim.y = 1;
        monarch_conv_bwd_cuda_kernel<32, 1, 1024, 2, false, 1, 8><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      } else {
        gridDim.x = B;
        gridDim.y = H;

        dk_f_out = torch::empty({B, H, fftsize + 1, 2}, x.options());

        blockDim.x = 32;
        blockDim.y = 1;
        monarch_conv_bwd_cuda_kernel<32, 1, 1024, 2, false, 1, 1><<<gridDim, blockDim, 8 * fftsize * sizeof(half)>>>(
            static_cast<at::BFloat16 *>(dout.data_ptr()),
            static_cast<at::BFloat16 *>(x.data_ptr()),
            static_cast<complex_bfloat16_t *>(k_f.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_fft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twid_r2r.data_ptr()),
            static_cast<complex_bfloat16_t *>(f_sqrt_N_ifft.data_ptr()),
            static_cast<complex_bfloat16_t *>(twiddle_factors_ifft.data_ptr()),
            static_cast<at::BFloat16 *>(dx_out.data_ptr()),
            static_cast<complex_bfloat16_t *>(dk_f_out.data_ptr()),
            in_gate.has_value() ? static_cast<at::BFloat16 *>(in_gate.value().data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(out_gate.value().data_ptr()) : nullptr,
            in_gate.has_value() ? static_cast<at::BFloat16 *>(din_gate.data_ptr()) : nullptr,
            out_gate.has_value() ? static_cast<at::BFloat16 *>(dout_gate.data_ptr()) : nullptr,
            B,
            H,
            N,
            sqrt_N);
      }
      
      break;
     default:
        AT_ERROR("Monarch backward not implemented for this sequence length");
  }
  
  CHECK_LAST_CUDA_ERROR();
  if (in_gate.has_value() && out_gate.has_value()) {
    return {dx_out, dk_f_out.sum(0), din_gate, dout_gate};
  } else if (in_gate.has_value()) {
    return {dx_out, dk_f_out.sum(0), din_gate};
  } else if (out_gate.has_value()) {
    return {dx_out, dk_f_out.sum(0), dout_gate};
  } else{
    return {dx_out, dk_f_out.sum(0)};
  }
}