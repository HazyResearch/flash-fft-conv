// Copyright (c) 2023 Dan Fu, Hermann Kumbong

#include <torch/extension.h>

#include <vector>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include "kernels_fp16/monarch_cuda_shared.h"
#include "kernels_fp16/monarch_cuda_16_16_16_bwd_complex_kernel.h"
#include "kernels_fp16/monarch_cuda_32_16_16_bwd_complex_kernel.h"
#include "kernels_fp16/monarch_cuda_16_32_32_bwd_complex_kernel.h"
#include "kernels_fp16/monarch_cuda_32_32_32_bwd_complex_kernel.h"
using namespace nvcuda;

// *************** FOR ERROR CHECKING *******************
#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL( call )                                                                                           \
    {                                                                                                                  \
        auto status = static_cast<hipError_t>( call );                                                                \
        if ( status != hipSuccess )                                                                                   \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                     "with "                                                                                           \
                     "%s (%d).\n",                                                                                     \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     hipGetErrorString( status ),                                                                     \
                     status );                                                                                         \
    }
#endif  // CUDA_RT_CALL
// *************** FOR ERROR CHECKING *******************

#ifndef CUDA_CHECK_ERROR
// Define some error checking macros.
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}
#endif  // CUDA_CHECK_ERROR

#ifndef CHECK_LAST_CUDA_ERROR
#define CHECK_LAST_CUDA_ERROR() checkLastBF16BwdComplex(__FILE__, __LINE__)
void checkLastBF16BwdComplex(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}
#endif  // CHECK_LAST_CUDA_ERROR

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
monarch_conv_bwd_cuda_16_16_16_complex(
  torch::Tensor dout_real,
  torch::Tensor dout_imag,
  torch::Tensor x_real,
  torch::Tensor x_imag,
  torch::Tensor k_f,
  torch::Tensor f_16_fft,
  torch::Tensor twiddle_factors_256_fft,
  torch::Tensor twiddle_factors_16_fft,
  torch::Tensor f_16_ifft,
  torch::Tensor twiddle_factors_256_ifft,
  torch::Tensor twiddle_factors_16_ifft,
  uint fftsize,
  uint N
){

  uint B = x_real.size(0);
  uint H = x_real.size(1);
  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // printf("gridDim.x = %d, gridDim.y = %d\n", gridDim.x, gridDim.y);
  torch::Tensor dx_out_real = torch::empty({B, H, N}, x_real.options());
  torch::Tensor dx_out_imag = torch::empty({B, H, N}, x_imag.options());

  torch::Tensor dk_f_out;

  switch (fftsize) {
  case 4096:
    if (B >= 4 && (B % 4) == 0 && (H % 8) == 0) {
    // if (true) {
      gridDim.x = B / 4;
      gridDim.y = H / 8;

      blockDim.x = 32;
      blockDim.y = 8;

      dk_f_out = torch::empty({B / 4, H, fftsize, 2}, x_real.options());
      monarch_conv_bwd_cuda_complex_kernel<32, 8, 4096, 1, 16, false, 4, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 256) * sizeof(half)>>>(
        static_cast<at::Half *>(dout_real.data_ptr()),
        static_cast<at::Half *>(dout_imag.data_ptr()),  
        static_cast<at::Half *>(x_real.data_ptr()),
        static_cast<at::Half *>(x_imag.data_ptr()),
        static_cast<complex_half_t *>(k_f.data_ptr()),
        static_cast<complex_half_t *>(f_16_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
        static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
        static_cast<at::Half *>(dx_out_real.data_ptr()),
        static_cast<at::Half *>(dx_out_imag.data_ptr()),
        static_cast<complex_half_t *>(dk_f_out.data_ptr()),
        B,
        H,
        N,
        16);
    }
    else if (B == 2 && (B % 2) == 0 && (H % 8) == 0) {
      gridDim.x = B / 2;
      gridDim.y = H / 8;

      blockDim.x = 32;
      blockDim.y = 8;

      dk_f_out = torch::empty({B / 2, H, fftsize, 2}, x_real.options());
      monarch_conv_bwd_cuda_complex_kernel<32, 8, 4096, 1, 16, false, 2, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 256) * sizeof(half)>>>(
        static_cast<at::Half *>(dout_real.data_ptr()),
        static_cast<at::Half *>(dout_imag.data_ptr()),  
        static_cast<at::Half *>(x_real.data_ptr()),
        static_cast<at::Half *>(x_imag.data_ptr()),
        static_cast<complex_half_t *>(k_f.data_ptr()),
        static_cast<complex_half_t *>(f_16_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
        static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
        static_cast<at::Half *>(dx_out_real.data_ptr()),
        static_cast<at::Half *>(dx_out_imag.data_ptr()),
        static_cast<complex_half_t *>(dk_f_out.data_ptr()),
        B,
        H,
        N,
        16);
    } else if ((H % 8) == 0) {
      gridDim.x = B;
      gridDim.y = H / 8;

      blockDim.x = 32;
      blockDim.y = 8;

      dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());
      monarch_conv_bwd_cuda_complex_kernel<32, 8, 4096, 1, 16, false, 1, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 256) * sizeof(half)>>>(
        static_cast<at::Half *>(dout_real.data_ptr()),
        static_cast<at::Half *>(dout_imag.data_ptr()),  
        static_cast<at::Half *>(x_real.data_ptr()),
        static_cast<at::Half *>(x_imag.data_ptr()),
        static_cast<complex_half_t *>(k_f.data_ptr()),
        static_cast<complex_half_t *>(f_16_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
        static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
        static_cast<at::Half *>(dx_out_real.data_ptr()),
        static_cast<at::Half *>(dx_out_imag.data_ptr()),
        static_cast<complex_half_t *>(dk_f_out.data_ptr()),
        B,
        H,
        N,
        16);
    } else {
      gridDim.x = B;
      gridDim.y = H;

      blockDim.x = 32;
      blockDim.y = 8;

      dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());
      monarch_conv_bwd_cuda_complex_kernel<32, 8, 4096, 1, 16, false, 1, 1, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 256) * sizeof(half)>>>(
        static_cast<at::Half *>(dout_real.data_ptr()),
        static_cast<at::Half *>(dout_imag.data_ptr()),  
        static_cast<at::Half *>(x_real.data_ptr()),
        static_cast<at::Half *>(x_imag.data_ptr()),
        static_cast<complex_half_t *>(k_f.data_ptr()),
        static_cast<complex_half_t *>(f_16_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_fft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
        static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_256_ifft.data_ptr()),
        static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
        static_cast<at::Half *>(dx_out_real.data_ptr()),
        static_cast<at::Half *>(dx_out_imag.data_ptr()),
        static_cast<complex_half_t *>(dk_f_out.data_ptr()),
        B,
        H,
        N,
        16);
    }
    break;
    default:
      AT_ERROR("Monarch backward not implemented for this sequence length");
  }

  CHECK_LAST_CUDA_ERROR();
  return std::make_tuple(dx_out_real, dx_out_imag, dk_f_out.sum(/*dim=*/0));
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
monarch_conv_bwd_cuda_32_16_16_complex(
  torch::Tensor dout_real,
  torch::Tensor dout_imag,
  torch::Tensor x_real,
  torch::Tensor x_imag,
  torch::Tensor k_f,
  torch::Tensor f_32_fft,
  torch::Tensor f_16_fft,
  torch::Tensor twiddle_factors_N_fft,
  torch::Tensor twiddle_factors_16_fft,
  torch::Tensor f_32_ifft,
  torch::Tensor f_16_ifft,
  torch::Tensor twiddle_factors_N_ifft,
  torch::Tensor twiddle_factors_16_ifft,
  uint fftsize,
  uint N
){

  uint B = x_real.size(0);
  uint H = x_real.size(1);
  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // printf("gridDim.x = %d, gridDim.y = %d\n", gridDim.x, gridDim.y);
  torch::Tensor dx_out_real = torch::empty({B, H, N}, x_real.options());
  torch::Tensor dx_out_imag = torch::empty({B, H, N}, x_imag.options());

  torch::Tensor dk_f_out;

  switch (fftsize) {
    case 8192:
      if (B >= 4 && (B % 4) == 0 && (H % 8) == 0) {
      // if (true) {
        gridDim.x = B / 4;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B / 4, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_complex_kernel<32), 8, 8192, 2, 1, 16, false, 4, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 102400));
        monarch_conv_bwd_cuda_complex_kernel<32, 8, 8192, 2, 1, 16, false, 4, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      }
      else if ((H % 8) == 0) {
        gridDim.x = B;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_complex_kernel<32), 8, 8192, 2, 1, 16, false, 1, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 102400));
        monarch_conv_bwd_cuda_complex_kernel<32, 8, 8192, 2, 1, 16, false, 1, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      } else {
        gridDim.x = B;
        gridDim.y = H;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_complex_kernel<32), 8, 8192, 2, 1, 16, false, 1, 1, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 102400));
        monarch_conv_bwd_cuda_complex_kernel<32, 8, 8192, 2, 1, 16, false, 1, 1, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_16_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      }
      
      break;
    default:
      AT_ERROR("Monarch backward not implemented for this sequence length");
  }
  
  CHECK_LAST_CUDA_ERROR();
  return std::make_tuple(dx_out_real, dx_out_imag, dk_f_out.sum(/*dim=*/0));
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
monarch_conv_bwd_cuda_16_32_32_complex(
  torch::Tensor dout_real,
  torch::Tensor dout_imag,
  torch::Tensor x_real,
  torch::Tensor x_imag,
  torch::Tensor k_f,
  torch::Tensor f_16_fft,
  torch::Tensor f_32_fft,
  torch::Tensor twiddle_factors_N_fft,
  torch::Tensor twiddle_factors_32_fft,
  torch::Tensor f_16_ifft,
  torch::Tensor f_32_ifft,
  torch::Tensor twiddle_factors_N_ifft,
  torch::Tensor twiddle_factors_32_ifft,
  uint fftsize,
  uint N
){

  uint B = x_real.size(0);
  uint H = x_real.size(1);
  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // printf("gridDim.x = %d, gridDim.y = %d\n", gridDim.x, gridDim.y);
  torch::Tensor dx_out_real = torch::empty({B, H, N}, x_real.options());
  torch::Tensor dx_out_imag = torch::empty({B, H, N}, x_imag.options());

  torch::Tensor dk_f_out;

  switch (fftsize) {
    case 16384:
      if (B >= 8 && (B % 8) == 0 && (H % 8) == 0) {
      // if (true) {
        gridDim.x = B / 8;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B / 8, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_16_32_32_complex_kernel<32), 8, 16384, 1, 2, 16, false, 8, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 140000));
        
        monarch_conv_bwd_cuda_16_32_32_complex_kernel<32, 8, 16384, 1, 2, 16, false, 8, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      }
      else if ((H % 8) == 0) {
        gridDim.x = B;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_16_32_32_complex_kernel<32), 8, 16384, 1, 2, 16, false, 1, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 140000));
        
        monarch_conv_bwd_cuda_16_32_32_complex_kernel<32, 8, 16384, 1, 2, 16, false, 1, 8, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      } else {
        gridDim.x = B;
        gridDim.y = H;

        blockDim.x = 32;
        blockDim.y = 8;
        
        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_16_32_32_complex_kernel<32), 8, 16384, 1, 2, 16, false, 1, 1, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 140000));
        
        monarch_conv_bwd_cuda_16_32_32_complex_kernel<32, 8, 16384, 1, 2, 16, false, 1, 1, 8><<<gridDim, blockDim, (4 * fftsize + 4 * 1024) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_16_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_16_ifft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      }
      
      break;
    default:
      AT_ERROR("Monarch backward not implemented for this sequence length");
  }
  
  CHECK_LAST_CUDA_ERROR();
  return std::make_tuple(dx_out_real, dx_out_imag, dk_f_out.sum(/*dim=*/0));
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
monarch_conv_bwd_cuda_32_32_32_complex(
  torch::Tensor dout_real,
  torch::Tensor dout_imag,
  torch::Tensor x_real,
  torch::Tensor x_imag,
  torch::Tensor k_f,
  torch::Tensor f_32_fft,
  torch::Tensor twiddle_factors_N_fft,
  torch::Tensor twiddle_factors_32_fft,
  torch::Tensor f_32_ifft,
  torch::Tensor twiddle_factors_N_ifft,
  torch::Tensor twiddle_factors_32_ifft,
  uint fftsize,
  uint N
){

  uint B = x_real.size(0);
  uint H = x_real.size(1);
  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // printf("gridDim.x = %d, gridDim.y = %d\n", gridDim.x, gridDim.y);
  torch::Tensor dx_out_real = torch::empty({B, H, N}, x_real.options());
  torch::Tensor dx_out_imag = torch::empty({B, H, N}, x_imag.options());

  torch::Tensor dk_f_out;

  switch (fftsize) {
    case 32768:
      if (B >= 8 && (B % 8) == 0 && (H % 8) == 0) {
        gridDim.x = B / 8;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;

        dk_f_out = torch::empty({B / 8, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_32_32_32_complex_kernel<32), 8, 32768, 2, 16, false, 8, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 135168));

        monarch_conv_bwd_cuda_32_32_32_complex_kernel<32, 8, 32768, 2, 16, false, 8, 8, 8><<<gridDim, blockDim, (2 * fftsize) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      } else if ((H % 8) == 0) {
        gridDim.x = B;
        gridDim.y = H / 8;

        blockDim.x = 32;
        blockDim.y = 8;

        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_32_32_32_complex_kernel<32), 8, 32768, 2, 16, false, 1, 8, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 135168));

        monarch_conv_bwd_cuda_32_32_32_complex_kernel<32, 8, 32768, 2, 16, false, 1, 8, 8><<<gridDim, blockDim, (2 * fftsize) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      } else {
        gridDim.x = B;
        gridDim.y = H;

        blockDim.x = 32;
        blockDim.y = 8;

        dk_f_out = torch::empty({B, H, fftsize, 2}, x_real.options());

        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(&monarch_conv_bwd_cuda_32_32_32_complex_kernel<32), 8, 32768, 2, 16, false, 1, 1, 8>, hipFuncAttributeMaxDynamicSharedMemorySize, 135168));

        monarch_conv_bwd_cuda_32_32_32_complex_kernel<32, 8, 32768, 2, 16, false, 1, 1, 8><<<gridDim, blockDim, (2 * fftsize) * sizeof(half)>>>(
            static_cast<at::Half *>(dout_real.data_ptr()),
            static_cast<at::Half *>(dout_imag.data_ptr()),  
            static_cast<at::Half *>(x_real.data_ptr()),
            static_cast<at::Half *>(x_imag.data_ptr()),
            static_cast<complex_half_t *>(k_f.data_ptr()),
            static_cast<complex_half_t *>(f_32_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_fft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_fft.data_ptr()),
            static_cast<complex_half_t *>(f_32_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_N_ifft.data_ptr()),
            static_cast<complex_half_t *>(twiddle_factors_32_ifft.data_ptr()),
            static_cast<at::Half *>(dx_out_real.data_ptr()),
            static_cast<at::Half *>(dx_out_imag.data_ptr()),
            static_cast<complex_half_t *>(dk_f_out.data_ptr()),
            B,
            H,
            N);
      }
      
      break;
    default:
      AT_ERROR("Monarch backward not implemented for this sequence length");
  }
  
  CHECK_LAST_CUDA_ERROR();
  return std::make_tuple(dx_out_real, dx_out_imag, dk_f_out.sum(/*dim=*/0));
}