#include "hip/hip_runtime.h"
// Copyright (c) 2023 Dan Fu, Hermann Kumbong

#include <torch/extension.h>

#include <vector>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include "shared.h"

using namespace nvcuda;

__global__ void butterfly_cuda_kernel_64(
    const __half2 *__restrict__ x,
    const __half2 *__restrict__ x_gate,
    const complex_half_t *__restrict__ d_f,
    const __half2 *__restrict__ twiddle_factors_real,
    const __half2 *__restrict__ twiddle_factors_imag,
    __half2 *__restrict__ out_real,
    __half2 *__restrict__ out_imag,
    uint B,
    uint H,
    int N)
{
    const int offset = blockIdx.y * H * 64 * 32 * gridDim.x + blockIdx.z * 16 * 64 * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
    const int tw_offset = blockIdx.x * 32 + threadIdx.x;
    int idx;
    int shared_offset;
    const int B_Y = blockDim.y;
    const int n = N / B_Y;
    

    extern __shared__ half x_shared[];
    half *d_f_real = &x_shared[N * N];
    half *d_f_imag = &d_f_real[N * N];
    half *twiddles_real_shared = &d_f_imag[N * N];
    half *twiddles_imag_shared = &twiddles_real_shared[N * N];
    half *out_real_shared = &twiddles_imag_shared[N * N];
    half *out_imag_shared = &out_real_shared[N * N];

    // #pragma unroll
    for (int i = 0; i < n; i++)
    {
        idx = (threadIdx.y + i * B_Y) * 32 * gridDim.x;
        shared_offset = (threadIdx.y + i * B_Y) * 32 + threadIdx.x;
        reinterpret_cast<__half2 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[tw_offset + idx];
        reinterpret_cast<__half2 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[tw_offset + idx];

        // #pragma unroll
        shared_offset = (threadIdx.y + i * B_Y) * 64 + threadIdx.x;
        d_f_real[shared_offset] = d_f[shared_offset].real();
        d_f_imag[shared_offset] = d_f[shared_offset].imag();

        d_f_real[shared_offset + blockDim.x] = d_f[shared_offset + blockDim.x].real();
        d_f_imag[shared_offset + blockDim.x] = d_f[shared_offset + blockDim.x].imag();
    }

    __half2 tmp_real, tmp_imag;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_real[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_real[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_imag[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_imag[4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[4][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_real[4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_imag[4];

    __syncthreads();

    for (int i = 0; i < 4; i++)
    {
        wmma::load_matrix_sync(a_frag_real[i], d_f_real + i * N * 16 + threadIdx.y * 16, N);
        wmma::load_matrix_sync(a_frag_imag[i], d_f_imag + i * N * 16 + threadIdx.y * 16, N);
        wmma::load_matrix_sync(tw_frag_real[i], twiddles_real_shared + threadIdx.y * N * 16 + i * 16, N);
        wmma::load_matrix_sync(tw_frag_imag[i], twiddles_imag_shared + threadIdx.y * N * 16 + i * 16, N);
    }

    for (int t = 0; t < 16; t++)
    {

        for (int i = 0; i < n; i++)
        {
            idx = (threadIdx.y + i * B_Y) * 32 * gridDim.x + t * 64 * 32 * gridDim.x;
            shared_offset = (threadIdx.y + i * B_Y) * 32 + threadIdx.x;
            if(x_gate != nullptr){
                reinterpret_cast<__half2 *>(x_shared)[shared_offset] = __hmul2(x[idx + offset], x_gate[idx + offset]);
            }else{
                reinterpret_cast<__half2 *>(x_shared)[shared_offset] = x[idx + offset];
            }
        }

        __syncthreads();

        for (int i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                wmma::load_matrix_sync(b_frag[i][j], x_shared + i * N * 16 + j * 16, N);
            }
        }

#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            wmma::fill_fragment(acc_frag_real[j], __float2half(0.0f));

            for (int k = 0; k < 4; k++)
            {
                wmma::mma_sync(acc_frag_real[j], a_frag_real[k], b_frag[k][j], acc_frag_real[j]);
            }
        }

#pragma unroll

        for (int j = 0; j < 4; j++)
        {
            wmma::fill_fragment(acc_frag_imag[j], __float2half(0.0f));

            for (int k = 0; k < 4; k++)
            {
                wmma::mma_sync(acc_frag_imag[j], a_frag_imag[k], b_frag[k][j], acc_frag_imag[j]);
            }
        }

#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            for (int k = 0; k < acc_frag_real[j].num_elements / 2; k++)
            {
                tmp_real = reinterpret_cast<__half2 *>(acc_frag_real[j].x)[k];
                tmp_imag = reinterpret_cast<__half2 *>(acc_frag_imag[j].x)[k];
                reinterpret_cast<__half2 *>(acc_frag_real[j].x)[k] = __hsub2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_real[j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_imag[j].x)[k]));
                reinterpret_cast<__half2 *>(acc_frag_imag[j].x)[k] = __hadd2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_imag[j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_real[j].x)[k]));
            }

            wmma::store_matrix_sync(out_real_shared + threadIdx.y * N * 16 + j * 16, acc_frag_real[j], N, wmma::mem_row_major);
            wmma::store_matrix_sync(out_imag_shared + threadIdx.y * N * 16 + j * 16, acc_frag_imag[j], N, wmma::mem_row_major);
        }

        __syncthreads();

#pragma unroll
        for (int i = 0; i < n; i++)
        {
            idx = offset + (threadIdx.y + i * B_Y) * 32 * gridDim.x + t * 64 * 32 * gridDim.x;
            out_real[idx] = reinterpret_cast<__half2 *>(out_real_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
            out_imag[idx] = reinterpret_cast<__half2 *>(out_imag_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
        }

        __syncthreads();
    }
}

__global__ void butterfly_cuda_kernel_32(
    const __half2 *__restrict__ x,
    const __half2 *__restrict__ x_gate,
    const complex_half_t *__restrict__ d_f,
    const __half2 *__restrict__ twiddle_factors_real,
    const __half2 *__restrict__ twiddle_factors_imag,
    __half2 *__restrict__ out_real,
    __half2 *__restrict__ out_imag,
    uint B,
    uint H,
    int N)
{
    const int offset = blockIdx.y * H * 32 * 32 * gridDim.x + blockIdx.z * 32 * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
    const int tw_offset = blockIdx.x * 32 + threadIdx.x;
    int idx;
    
    int shared_offset;
    const int B_Y = blockDim.y;
    const int n = N / B_Y;
    

    __shared__ half x_shared[32 * 64];
    __shared__ half d_f_real[32 * 32];
    __shared__ half d_f_imag[32 * 32];
    __shared__ half twiddles_real_shared[32 * 64];
    __shared__ half twiddles_imag_shared[32 * 64];
    __shared__ half out_real_shared[32 * 64];
    __shared__ half out_imag_shared[32 * 64];

    // #pragma unroll
    for (int i = 0; i < n; i++)
    {
        idx = (threadIdx.y + i * B_Y) * 32 * gridDim.x;
        shared_offset = (threadIdx.y + i * B_Y) * 32 + threadIdx.x;
        if(x_gate == nullptr){
            reinterpret_cast<__half2 *>(x_shared)[shared_offset] = x[idx + offset];
        }else{
            reinterpret_cast<__half2 *>(x_shared)[shared_offset] = __hmul2(x[idx + offset], x_gate[idx + offset]);
        }
        reinterpret_cast<__half2 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[tw_offset + idx];
        reinterpret_cast<__half2 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[tw_offset + idx];

        // #pragma unroll
        d_f_real[shared_offset] = d_f[shared_offset].real();
        d_f_imag[shared_offset] = d_f[shared_offset].imag();
    }

    __syncthreads();

    if (threadIdx.y < N / 16)
    {
        __half2 tmp_real, tmp_imag;

        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_real[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_real[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_imag[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_imag[2][2];
        wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[2][2];
        wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_real[2][2];
        wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_imag[2][2];

        int t = threadIdx.y * 32;

        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::load_matrix_sync(a_frag_real[i][j], d_f_real + j * N * 16 + i * 16, N);
                wmma::load_matrix_sync(a_frag_imag[i][j], d_f_imag + j * N * 16 + i * 16, N);
                wmma::load_matrix_sync(b_frag[i][j], x_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
                wmma::load_matrix_sync(tw_frag_real[i][j], twiddles_real_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
                wmma::load_matrix_sync(tw_frag_imag[i][j], twiddles_imag_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::fill_fragment(acc_frag_real[i][j], __float2half(0.0f));

                for (int k = 0; k < 2; k++)
                {
                    wmma::mma_sync(acc_frag_real[i][j], a_frag_real[i][k], b_frag[k][j], acc_frag_real[i][j]);
                }
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::fill_fragment(acc_frag_imag[i][j], __float2half(0.0f));

                for (int k = 0; k < 2; k++)
                {
                    wmma::mma_sync(acc_frag_imag[i][j], a_frag_imag[i][k], b_frag[k][j], acc_frag_imag[i][j]);
                }
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                for (int k = 0; k < acc_frag_real[i][j].num_elements / 2; k++)
                {
                    tmp_real = reinterpret_cast<__half2 *>(acc_frag_real[i][j].x)[k];
                    tmp_imag = reinterpret_cast<__half2 *>(acc_frag_imag[i][j].x)[k];
                    reinterpret_cast<__half2 *>(acc_frag_real[i][j].x)[k] = __hsub2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_real[i][j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_imag[i][j].x)[k]));
                    reinterpret_cast<__half2 *>(acc_frag_imag[i][j].x)[k] = __hadd2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_imag[i][j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_real[i][j].x)[k]));
                }

                wmma::store_matrix_sync(out_real_shared + i * 2 * N * 16 + j * 16 + t, acc_frag_real[i][j], 2 * N, wmma::mem_row_major);
                wmma::store_matrix_sync(out_imag_shared + i * 2 * N * 16 + j * 16 + t, acc_frag_imag[i][j], 2 * N, wmma::mem_row_major);
            }
        }
    }

    __syncthreads();

#pragma unroll
    for (int i = 0; i < n; i++)
    {
        idx = offset + (threadIdx.y + i * B_Y) * 32 * gridDim.x;
        out_real[idx] = reinterpret_cast<__half2 *>(out_real_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
        out_imag[idx] = reinterpret_cast<__half2 *>(out_imag_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
    }
}

__global__ void butterfly_cuda_kernel_128(
    const __half2 *__restrict__ x,
    const __half2 *__restrict__ x_gate,
    const complex_half_t *__restrict__ d_f,
    const __half2 *__restrict__ twiddle_factors_real,
    const __half2 *__restrict__ twiddle_factors_imag,
    __half2 *__restrict__ out_real,
    __half2 *__restrict__ out_imag,
    uint B,
    uint H,
    int N)
{
    const int offset = blockIdx.y * H * 128 * 32 * gridDim.x * 2 + blockIdx.z * 16 * 128 * 32 * gridDim.x * 2 + blockIdx.x * 64 + threadIdx.x;
    const int tw_offset = blockIdx.x * 64 + threadIdx.x;
    int idx;
    
    int shared_offset;
    const int B_Y = blockDim.y;
    const int n = N / B_Y;
    

    extern __shared__ half shared_real[];
    half *shared_imag = &shared_real[128 * 128];


    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_real[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_real[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_imag[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_imag[8];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[8][8];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_real[8];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_imag[8];

    for (int i = 0; i < n; i++)
    {
        for(int j=0; j< 4; j++){
            shared_offset = (threadIdx.y + i * B_Y) * 128 + threadIdx.x + j * blockDim.x;
            shared_real[shared_offset] = d_f[shared_offset].real();
            shared_imag[shared_offset] = d_f[shared_offset].imag();
        }
    }

    __syncthreads();


    for (int i = 0; i < 8; i++){
        wmma::load_matrix_sync(a_frag_real[i], shared_real + i * 128 * 16 + threadIdx.y * 16, 128);
        wmma::load_matrix_sync(a_frag_imag[i], shared_imag + i * 128 * 16 + threadIdx.y * 16, 128);
    }


    __syncthreads();



    for (int i = 0; i < n; i++)
    {
        for(int j=0; j< 2; j++){
            idx = (threadIdx.y + i * B_Y) * 32 * 2 * gridDim.x + j * blockDim.x;
            shared_offset = (threadIdx.y + i * B_Y) * 64 + threadIdx.x + j * blockDim.x;   
            reinterpret_cast<__half2*>(shared_real)[shared_offset] = twiddle_factors_real[tw_offset + idx];
            reinterpret_cast<__half2*>(shared_imag)[shared_offset] = twiddle_factors_imag[tw_offset + idx];
        }
    }

    __syncthreads();


    for (int i = 0; i < 8; i++){
        wmma::load_matrix_sync(tw_frag_real[i], shared_real + threadIdx.y * 128 * 16 + i * 16, 128);
        wmma::load_matrix_sync(tw_frag_imag[i], shared_imag + threadIdx.y * 128 * 16 + i * 16, 128);
    }

    __syncthreads();


    for(int t=0; t< 16; t++){
        for (int i = 0; i < n; i++)
        {
            for(int j=0; j< 2; j++){
                idx = (threadIdx.y + i * B_Y) * 32 * 2 * gridDim.x + j * blockDim.x + t * 128 * 32 * 2 * gridDim.x;
                shared_offset = (threadIdx.y + i * B_Y) * 64 + threadIdx.x + j * blockDim.x;
                if(x_gate != nullptr){   
                    reinterpret_cast<__half2*>(shared_real)[shared_offset] = __hmul2(x[idx + offset], x_gate[idx + offset]);
                }else{
                    reinterpret_cast<__half2*>(shared_real)[shared_offset] = x[offset + idx];
                }

            }
        }


        __syncthreads();


        for (int i = 0; i < 8; i++)
        {
            for (int j = 0; j < 8; j++)
            {
                wmma::load_matrix_sync(b_frag[i][j], shared_real + i * 128 * 16 + j * 16, 128);
            }
        }

        __syncthreads();

        #pragma unroll
            for (int j = 0; j < 8; j++)
            {
                wmma::fill_fragment(acc_frag_real[j], __float2half(0.0f));

                for (int k = 0; k < 8; k++)
                {
                    wmma::mma_sync(acc_frag_real[j], a_frag_real[k], b_frag[k][j], acc_frag_real[j]);
                }
            }

    #pragma unroll

            for (int j = 0; j < 8; j++)
            {
                wmma::fill_fragment(acc_frag_imag[j], __float2half(0.0f));

                for (int k = 0; k < 8; k++)
                {
                    wmma::mma_sync(acc_frag_imag[j], a_frag_imag[k], b_frag[k][j], acc_frag_imag[j]);
                }
            }

            __half2 tmp_real, tmp_imag;
    #pragma unroll
            for (int j = 0; j < 8; j++)
            {
                for (int k = 0; k < acc_frag_real[j].num_elements / 2; k++)
                {
                    tmp_real = reinterpret_cast<__half2 *>(acc_frag_real[j].x)[k];
                    tmp_imag = reinterpret_cast<__half2 *>(acc_frag_imag[j].x)[k];
                    reinterpret_cast<__half2 *>(acc_frag_real[j].x)[k] = __hsub2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_real[j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_imag[j].x)[k]));
                    reinterpret_cast<__half2 *>(acc_frag_imag[j].x)[k] = __hadd2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_imag[j].x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_real[j].x)[k]));
                }

                wmma::store_matrix_sync(shared_real + threadIdx.y * 128 * 16 + j * 16, acc_frag_real[j], 128, wmma::mem_row_major);
                wmma::store_matrix_sync(shared_imag + threadIdx.y * 128 * 16 + j * 16, acc_frag_imag[j], 128, wmma::mem_row_major);
            }

            __syncthreads();

    #pragma unroll
            for (int i = 0; i < n; i++)
            {
                for(int j=0; j< 2; j++){
                    idx =  (threadIdx.y + i * B_Y) * 32 * 2 * gridDim.x + j * blockDim.x + t * 128 * 32 * 2 * gridDim.x;
                    shared_offset = (threadIdx.y + i * B_Y) * 64 + threadIdx.x + j * blockDim.x;
                    out_real[offset + idx] = reinterpret_cast<__half2*>(shared_real)[shared_offset];
                    out_imag[offset + idx] = reinterpret_cast<__half2*>(shared_imag)[shared_offset];
                }
            }

            __syncthreads();
    }
}


__global__ void butterfly_cuda_kernel_16(
    const __half2 *__restrict__ x,
    const __half2 *__restrict__ x_gate,
    const complex_half_t *__restrict__ d_f,
    const __half2 *__restrict__ twiddle_factors_real,
    const __half2 *__restrict__ twiddle_factors_imag,
    __half2 *__restrict__ out_real,
    __half2 *__restrict__ out_imag,
    uint B,
    uint H,
    int N)
{
    const int offset = blockIdx.y * H * 16 * 32 * gridDim.x + blockIdx.z * 16 * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
    const int tw_offset = blockIdx.x * 32 + threadIdx.x;
    int idx;
    
    int shared_offset;
    const int B_Y = blockDim.y;
    const int n = N / B_Y;
    

    __shared__ half x_shared[16 * 64];
    __shared__ half d_f_real[16 * 16];
    __shared__ half d_f_imag[16 * 16];
    __shared__ half twiddles_real_shared[16 * 64];
    __shared__ half twiddles_imag_shared[16 * 64];
    __shared__ half out_real_shared[16 * 64];
    __shared__ half out_imag_shared[16 * 64];

    // #pragma unroll
    for (int i = 0; i < n; i++)
    {
        idx = (threadIdx.y + i * B_Y) * 32 * gridDim.x;
        shared_offset = (threadIdx.y + i * B_Y) * 32 + threadIdx.x;

        if(x_gate != NULL)
            reinterpret_cast<__half2 *>(x_shared)[shared_offset] = __hmul2(x[idx + offset], x_gate[idx + offset]);
        else
            reinterpret_cast<__half2 *>(x_shared)[shared_offset] = x[idx + offset];
        reinterpret_cast<__half2 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[tw_offset + idx];
        reinterpret_cast<__half2 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[tw_offset + idx];

        // #pragma unroll

        if(threadIdx.x  < 16 ){
            shared_offset = (threadIdx.y + i * B_Y) * 16 + threadIdx.x;
            d_f_real[shared_offset] = d_f[shared_offset].real();
            d_f_imag[shared_offset] = d_f[shared_offset].imag();
        }
    }

    __syncthreads();

    if (threadIdx.y < 4)
    {
        __half2 tmp_real, tmp_imag;

        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_real;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_real;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> tw_frag_imag;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag_imag;
        wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
        wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_real;
        wmma::fragment<wmma::accumulator, 16, 16, 16, half> acc_frag_imag;

        wmma::load_matrix_sync(a_frag_real, d_f_real, N);
        wmma::load_matrix_sync(a_frag_imag, d_f_imag, N);
        wmma::load_matrix_sync(b_frag, x_shared + threadIdx.y * 16, 64);
        wmma::load_matrix_sync(tw_frag_real, twiddles_real_shared + threadIdx.y * 16, 64);
        wmma::load_matrix_sync(tw_frag_imag, twiddles_imag_shared + threadIdx.y * 16, 64);


        wmma::fill_fragment(acc_frag_real, __float2half(0.0f));


        wmma::mma_sync(acc_frag_real, a_frag_real, b_frag, acc_frag_real);


        wmma::fill_fragment(acc_frag_imag, __float2half(0.0f));


        wmma::mma_sync(acc_frag_imag, a_frag_imag, b_frag, acc_frag_imag);



        for (int k = 0; k < acc_frag_real.num_elements / 2; k++)
        {
            tmp_real = reinterpret_cast<__half2 *>(acc_frag_real.x)[k];
            tmp_imag = reinterpret_cast<__half2 *>(acc_frag_imag.x)[k];
            reinterpret_cast<__half2 *>(acc_frag_real.x)[k] = __hsub2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_real.x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_imag.x)[k]));
            reinterpret_cast<__half2 *>(acc_frag_imag.x)[k] = __hadd2(__hmul2(tmp_real, reinterpret_cast<__half2 *>(tw_frag_imag.x)[k]), __hmul2(tmp_imag, reinterpret_cast<__half2 *>(tw_frag_real.x)[k]));
        }

        wmma::store_matrix_sync(out_real_shared + threadIdx.y * 16, acc_frag_real, 64, wmma::mem_row_major);
        wmma::store_matrix_sync(out_imag_shared + threadIdx.y * 16, acc_frag_imag, 64, wmma::mem_row_major);
    }

    __syncthreads();

#pragma unroll
    for (int i = 0; i < n; i++)
    {
        idx = offset + (threadIdx.y + i * B_Y) * 32 * gridDim.x;
        out_real[idx] = reinterpret_cast<__half2 *>(out_real_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
        out_imag[idx] = reinterpret_cast<__half2 *>(out_imag_shared)[(threadIdx.y + i * B_Y) * 32 + threadIdx.x];
    }
}


std::vector<torch::Tensor> butterfly_cuda(
    torch::Tensor x,
    torch::Tensor d_f,
    torch::Tensor twiddle_factors_real,
    torch::Tensor twiddle_factors_imag,
    std::optional<at::Tensor> x_gate = std::nullopt)
{

    uint B = x.size(0);
    uint H = x.size(1);
    // uint m = x.size(1);

    // const int TILE_SIZE = 16;
    uint N = x.size(2);
    uint M = x.size(3);
    dim3 gridDim;
    dim3 blockDim;

    gridDim.y = B;
    gridDim.z = H;

    torch::Tensor out_real = torch::empty({B, H, N, M}, x.options());
    torch::Tensor out_imag = torch::empty({B, H, N, M}, x.options());

    //set blockDims
    switch(N){
        case 128:
            blockDim.x = 32;
            blockDim.y = 8;
            break;
        default:
            blockDim.x = 32;
            blockDim.y = 4;
            break;
    }

    //set gridDim.x
    switch(N){
        case 128:
            switch (M){
                case 16384:
                    gridDim.x = 128;
                    break;
                case 8192:
                    gridDim.x = 64;
                    break;
                case 4096:
                    gridDim.x = 32;
                    break;
                default:
                    gridDim.x = 256;
                    break;
            }
            break;
        default:
            switch (M){
                case 16384:
                    gridDim.x = 256;
                    break;
                case 8192:
                    gridDim.x = 128;
                    break;
                case 4096:
                    gridDim.x = 64;
                    break;
                default:
                    gridDim.x = 512;
                    break;
            }
            break;
    }

    switch (N)
    {
    case 16:
        butterfly_cuda_kernel_16<<<gridDim, blockDim>>>(
            static_cast<__half2 *>(x.data_ptr()),
            x_gate ? static_cast<__half2 *>(x_gate.value().data_ptr()) : nullptr,
            static_cast<complex_half_t *>(d_f.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_real.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_imag.data_ptr()),
            static_cast<__half2 *>(out_real.data_ptr()),
            static_cast<__half2 *>(out_imag.data_ptr()),
            B,
            H,
            N);
        break;
    case 32:
        butterfly_cuda_kernel_32<<<gridDim, blockDim>>>(
            static_cast<__half2 *>(x.data_ptr()),
            x_gate ? static_cast<__half2 *>(x_gate.value().data_ptr()) : nullptr,
            static_cast<complex_half_t *>(d_f.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_real.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_imag.data_ptr()),
            static_cast<__half2 *>(out_real.data_ptr()),
            static_cast<__half2 *>(out_imag.data_ptr()),
            B,
            H,
            N);
        break;

    case 64:
        gridDim.z = H / 16;
        hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_64), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

        butterfly_cuda_kernel_64<<<gridDim, blockDim, 57344>>>(
            static_cast<__half2 *>(x.data_ptr()),
            x_gate ? static_cast<__half2 *>(x_gate.value().data_ptr()) : nullptr,
            static_cast<complex_half_t *>(d_f.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_real.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_imag.data_ptr()),
            static_cast<__half2 *>(out_real.data_ptr()),
            static_cast<__half2 *>(out_imag.data_ptr()),
            B,
            H,
            N);
        break;
    case 128:
        gridDim.z = H / 16;
        hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

        butterfly_cuda_kernel_128<<<gridDim, blockDim, 65536>>>(
            static_cast<__half2 *>(x.data_ptr()),
            x_gate ? static_cast<__half2 *>(x_gate.value().data_ptr()) : nullptr,
            static_cast<complex_half_t *>(d_f.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_real.data_ptr()),
            static_cast<__half2 *>(twiddle_factors_imag.data_ptr()),
            static_cast<__half2 *>(out_real.data_ptr()),
            static_cast<__half2 *>(out_imag.data_ptr()),
            B,
            H,
            N);
        break;

    default:
    printf("Not yet implemented \n");
        break;
    }

    return {out_real, out_imag};
}