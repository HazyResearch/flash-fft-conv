#include "hip/hip_runtime.h"
// Copyright (c) 2023 Dan Fu, Hermann Kumbong

#include <torch/extension.h>

#include <vector>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include "shared.h"

using namespace nvcuda;


template <int K>
__global__ void butterfly_cuda_kernel_64(
    const __hip_bfloat162 *__restrict__ x,
    const __hip_bfloat162 *__restrict__ x_gate,
    const __hip_bfloat162 *__restrict__ d_f_real,
    const __hip_bfloat162 *__restrict__ d_f_imag,
    const __hip_bfloat162 *__restrict__ twiddle_factors_real,
    const __hip_bfloat162 *__restrict__ twiddle_factors_imag,
    __hip_bfloat162 *__restrict__ out_real,
    __hip_bfloat162 *__restrict__ out_imag,
    uint B,
    uint H,
    int M)
{
    const int max_idx = M / 2; //actually should be -1 since indices are 0-based but we are using < instead of <=
    const int offset = blockIdx.y * H * M/2 + blockIdx.z * 16 *  M/2;
    const int out_offset = blockIdx.y * H * 64 * 32 * gridDim.x + blockIdx.z * 16 * 64 * 32 * gridDim.x;
    int idx;
    int t_offset;
    int out_t_offset;
    int shared_offset;
    const int N = 64;
    

    extern __shared__ __hip_bfloat16 x_shared[];
    __hip_bfloat16 *d_f_real_shared = &x_shared[K * 16 * N];
    __hip_bfloat16 *d_f_imag_shared = &d_f_real_shared[N * N];
    __hip_bfloat16 *twiddles_real_shared = &d_f_imag_shared[N * N];
    __hip_bfloat16 *twiddles_imag_shared = &twiddles_real_shared[N * N];
    float *out_real_shared = reinterpret_cast<float*>(&twiddles_imag_shared[N * N]);
    float *out_imag_shared = &out_real_shared[N * N];

    // #pragma unroll
    for (int i = threadIdx.y; i < N; i+=blockDim.y)
    {
        idx = i * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
        shared_offset = i * 32 + threadIdx.x;
        reinterpret_cast<__hip_bfloat162 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[idx];
        reinterpret_cast<__hip_bfloat162 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[idx];

        // #pragma unroll
        shared_offset = i * 32 + threadIdx.x;
        reinterpret_cast<__hip_bfloat162 *>(d_f_real_shared)[shared_offset] = d_f_real[shared_offset];
        reinterpret_cast<__hip_bfloat162 *>(d_f_imag_shared)[shared_offset] = d_f_imag[shared_offset];
    }

    float2 tmp_real, tmp_imag;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_real[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_real[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_imag[4];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_imag[4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> b_frag[4][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_real[4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_imag[4];

    __syncthreads();

    for (int i = 0; i < 4; i++)
    {
        wmma::load_matrix_sync(a_frag_real[i], d_f_real_shared + i * N * 16 + threadIdx.y * 16, N);
        wmma::load_matrix_sync(a_frag_imag[i], d_f_imag_shared + i * N * 16 + threadIdx.y * 16, N);
        wmma::load_matrix_sync(tw_frag_real[i], twiddles_real_shared + threadIdx.y * N * 16 + i * 16, N);
        wmma::load_matrix_sync(tw_frag_imag[i], twiddles_imag_shared + threadIdx.y * N * 16 + i * 16, N);
    }

    for (int t = 0; t < 16; t++)
    {
        t_offset = t * M/2;
        out_t_offset = t * 64 * 32 * gridDim.x;

        for (int i = threadIdx.y; i < N; i+=blockDim.y)
        {
            if(i < K * 16){
                idx = i * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
                shared_offset = i * 32 + threadIdx.x;
                if(x_gate != nullptr){
                    reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? __hmul2(x[idx + offset + t_offset], x_gate[idx + offset + t_offset]) : __floats2bfloat162_rn(0.0f, 0.0f);
                }else{
                    reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? x[idx + offset + t_offset] : __floats2bfloat162_rn(0.0f, 0.0f);
                }
            }
        }

        __syncthreads();

        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                wmma::load_matrix_sync(b_frag[i][j], x_shared + i * N * 16 + j * 16, N);
            }
        }

#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            wmma::fill_fragment(acc_frag_real[j], 0.0f);

            for (int k = 0; k < K; k++)
            {
                wmma::mma_sync(acc_frag_real[j], a_frag_real[k], b_frag[k][j], acc_frag_real[j]);
            }
        }

#pragma unroll

        for (int j = 0; j < 4; j++)
        {
            wmma::fill_fragment(acc_frag_imag[j], 0.0f);

            for (int k = 0; k < K; k++)
            {
                wmma::mma_sync(acc_frag_imag[j], a_frag_imag[k], b_frag[k][j], acc_frag_imag[j]);
            }
        }

#pragma unroll
        for (int j = 0; j < 4; j++)
        {
            for (int k = 0; k < acc_frag_real[j].num_elements / 2; k++)
            {
                tmp_real = reinterpret_cast<float2 *>(acc_frag_real[j].x)[k];
                tmp_imag = reinterpret_cast<float2 *>(acc_frag_imag[j].x)[k];
                
                reinterpret_cast<float2 *>(acc_frag_real[j].x)[k] = tmp_real * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[j].x)[k]) - tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[j].x)[k]);
                reinterpret_cast<float2 *>(acc_frag_imag[j].x)[k] = tmp_real * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[j].x)[k]) + tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[j].x)[k]);
            }

            wmma::store_matrix_sync(out_real_shared + threadIdx.y * N * 16 + j * 16, acc_frag_real[j], N, wmma::mem_row_major);
            wmma::store_matrix_sync(out_imag_shared + threadIdx.y * N * 16 + j * 16, acc_frag_imag[j], N, wmma::mem_row_major);
        }

        __syncthreads();

#pragma unroll
        for (int i = threadIdx.y; i < N; i+=blockDim.y)
        {
            idx = i * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
            shared_offset = i * 32 + threadIdx.x;
            out_real[out_offset + out_t_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_real_shared)[shared_offset]);
            out_imag[out_offset + out_t_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_imag_shared)[shared_offset]);
        }

        __syncthreads();
    }
}

template <int K>
__global__ void butterfly_cuda_kernel_32(
    const __hip_bfloat162 *__restrict__ x,
    const __hip_bfloat162 *__restrict__ x_gate,
    const __hip_bfloat16 *__restrict__ d_f_real,
    const __hip_bfloat16 *__restrict__ d_f_imag,
    const __hip_bfloat162 *__restrict__ twiddle_factors_real,
    const __hip_bfloat162 *__restrict__ twiddle_factors_imag,
    __hip_bfloat162 *__restrict__ out_real,
    __hip_bfloat162 *__restrict__ out_imag,
    uint B,
    uint H,
    int M)
{
    const int N  = 32;
    const int max_idx = M / 2; //actually should be -1 since indices are 0-based but we are using < instead of <=
   
    const int offset  =  blockIdx.y * H * M / 2 + blockIdx.z * M / 2; 
    const int out_offset = blockIdx.y * H * 32 * 32 * gridDim.x + blockIdx.z * 32 * 32 * gridDim.x;
    

    __shared__ __hip_bfloat16 x_shared[K * 16 * 64];
    __shared__ __hip_bfloat16 d_f_real_shared[32 * 32];
    __shared__ __hip_bfloat16 d_f_imag_shared[32 * 32];
    __shared__ __hip_bfloat16 twiddles_real_shared[32 * 64];
    __shared__ __hip_bfloat16 twiddles_imag_shared[32 * 64];
    __shared__ float out_real_shared[32 * 64];
    __shared__ float out_imag_shared[32 * 64];

    // #pragma unroll
    for (int i = threadIdx.y; i<32; i+=blockDim.y)
    {
        int idx = i * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
        int shared_offset = i * 32 + threadIdx.x;

        if(i < K * 16){
            if(x_gate != nullptr){
                reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? __hmul2(x[idx + offset], x_gate[idx + offset]) : __floats2bfloat162_rn(0.0f, 0.0f);
            }else{
                reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? x[idx + offset] : __floats2bfloat162_rn(0.0f, 0.0f);
            }
        }
        reinterpret_cast<__hip_bfloat162 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[idx];
        reinterpret_cast<__hip_bfloat162 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[idx];

        // #pragma unroll
        d_f_real_shared[shared_offset] = d_f_real[shared_offset];
        d_f_imag_shared[shared_offset] = d_f_imag[shared_offset];
    }

    __syncthreads();

    if (threadIdx.y < N / 16)
    {
        float2 tmp_real, tmp_imag;

        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_real[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_real[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_imag[2][2];
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_imag[2][2];
        wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> b_frag[K][2];
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_real[2][2];
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_imag[2][2];

        int t = threadIdx.y * 32;

        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::load_matrix_sync(a_frag_real[i][j], d_f_real_shared + j * N * 16 + i * 16, N);
                wmma::load_matrix_sync(a_frag_imag[i][j], d_f_imag_shared + j * N * 16 + i * 16, N);
                if(i < K){
                    wmma::load_matrix_sync(b_frag[i][j], x_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
                }
                wmma::load_matrix_sync(tw_frag_real[i][j], twiddles_real_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
                wmma::load_matrix_sync(tw_frag_imag[i][j], twiddles_imag_shared + i * 2 * N * 16 + j * 16 + t, 2 * N);
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::fill_fragment(acc_frag_real[i][j], 0.0f);

                for (int k = 0; k < K; k++)
                {
                    wmma::mma_sync(acc_frag_real[i][j], a_frag_real[i][k], b_frag[k][j], acc_frag_real[i][j]);
                }
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                wmma::fill_fragment(acc_frag_imag[i][j], 0.0f);

                for (int k = 0; k < K; k++)
                {
                    wmma::mma_sync(acc_frag_imag[i][j], a_frag_imag[i][k], b_frag[k][j], acc_frag_imag[i][j]);
                }
            }
        }

#pragma unroll
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                 for (int k = 0; k < acc_frag_real[i][j].num_elements / 2; k++)
                {
                    tmp_real = 	reinterpret_cast<float2 *>(acc_frag_real[i][j].x)[k];
                    tmp_imag = 	reinterpret_cast<float2 *>(acc_frag_imag[i][j].x)[k];
                    reinterpret_cast<float2 *>(acc_frag_real[i][j].x)[k] = 	tmp_real  * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[i][j].x)[k]) - tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[i][j].x)[k]);
                    reinterpret_cast<float2 *>(acc_frag_imag[i][j].x)[k] =  tmp_real  * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[i][j].x)[k]) + tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[i][j].x)[k]);
                }
                wmma::store_matrix_sync(out_real_shared + i * 2 * N * 16 + j * 16 + t, acc_frag_real[i][j], 2 * N, wmma::mem_row_major);
                wmma::store_matrix_sync(out_imag_shared + i * 2 * N * 16 + j * 16 + t, acc_frag_imag[i][j], 2 * N, wmma::mem_row_major);
            }
        }
    }

    __syncthreads();

#pragma unroll
    for (int i = threadIdx.y; i<32; i+=blockDim.y)
    {
        int idx = i * 32 * gridDim.x + blockIdx.x * 32 + threadIdx.x;
        out_real[out_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_real_shared)[i * 32 + threadIdx.x]);
        out_imag[out_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_imag_shared)[i * 32 + threadIdx.x]);
    }
}

template <int K>
__global__ void butterfly_cuda_kernel_128(
    const __hip_bfloat162 *__restrict__ x,
    const __hip_bfloat162 *__restrict__ x_gate,
    const __hip_bfloat162 *__restrict__ d_f_real,
    const __hip_bfloat162 *__restrict__ d_f_imag,
    const __hip_bfloat162 *__restrict__ twiddle_factors_real,
    const __hip_bfloat162 *__restrict__ twiddle_factors_imag,
    __hip_bfloat162 *__restrict__ out_real,
    __hip_bfloat162 *__restrict__ out_imag,
    uint B,
    uint H,
    int M)
{
    const int max_idx = M / 2; //actually should be -1 since indices are 0-based but we are using < instead of <=
    const int offset = blockIdx.y * H * M/2 + blockIdx.z * 16 *  M/2;
    const int out_offset = blockIdx.y * H * 128 * 32 * 2 * gridDim.x + blockIdx.z * 16 * 128 * 32 * 2 * gridDim.x;
    const int N = 128;
    int idx;
    int t_offset;
    int out_t_offset;
    int shared_offset;

    extern __shared__ __hip_bfloat16 shared_real[];
    __hip_bfloat16 *shared_imag = &shared_real[128 * 128];


    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_real[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_real[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_imag[8];
    wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_imag[8];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> b_frag[K][8];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_real[8];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_imag[8];

    for (int i = threadIdx.y ; i < N; i+=blockDim.y)
    {
        for(int j=0; j< 2; j++){
            shared_offset = i * 64 + threadIdx.x + j * blockDim.x;
            reinterpret_cast<__hip_bfloat162 *>(shared_real)[shared_offset] = d_f_real[shared_offset];
            reinterpret_cast<__hip_bfloat162 *>(shared_imag)[shared_offset] = d_f_imag[shared_offset];
        }
    }

    __syncthreads();


    for (int i = 0; i < 8; i++){
        wmma::load_matrix_sync(a_frag_real[i], shared_real + i * 128 * 16 + threadIdx.y * 16, 128);
        wmma::load_matrix_sync(a_frag_imag[i], shared_imag + i * 128 * 16 + threadIdx.y * 16, 128);
    }


    __syncthreads();



    for (int i = threadIdx.y; i < N; i+=blockDim.y)
    {
        for(int j=0; j< 2; j++){
            idx = i * 32 * 2 * gridDim.x + j * blockDim.x + blockIdx.x * 64 + threadIdx.x;
            shared_offset = i * 64 + threadIdx.x + j * blockDim.x; 
            reinterpret_cast<__hip_bfloat162*>(shared_real)[shared_offset] = twiddle_factors_real[idx];
            reinterpret_cast<__hip_bfloat162*>(shared_imag)[shared_offset] = twiddle_factors_imag[idx];
        }
    }

    __syncthreads();


    for (int i = 0; i < 8; i++){
        wmma::load_matrix_sync(tw_frag_real[i], shared_real + threadIdx.y * 128 * 16 + i * 16, 128);
        wmma::load_matrix_sync(tw_frag_imag[i], shared_imag + threadIdx.y * 128 * 16 + i * 16, 128);
    }

    __syncthreads();


    for(int t=0; t< 16; t++){
        t_offset = t * M/2;
        out_t_offset = t * 128 * 32 * 2 * gridDim.x;

        for (int i = threadIdx.y; i < N; i+=blockDim.y)
        {
            if(i < K * 16){
                for(int j=0; j< 2; j++){
                    idx = i * 32 * 2 * gridDim.x + j * blockDim.x + blockIdx.x * 64 + threadIdx.x;
                    shared_offset = i * 64 + threadIdx.x + j * blockDim.x;
                    if(x_gate != nullptr){
                        reinterpret_cast<__hip_bfloat162*>(shared_real)[shared_offset] = idx < max_idx ?  __hmul2(x[idx + offset + t_offset], x_gate[idx + offset + t_offset]) : __floats2bfloat162_rn(0.0f, 0.0f);
                    }else{  
                        reinterpret_cast<__hip_bfloat162*>(shared_real)[shared_offset] = idx < max_idx ? x[idx + offset + t_offset] : __floats2bfloat162_rn(0.0f, 0.0f);
                    }
                }
            }
        }


        __syncthreads();


        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < 8; j++)
            {
                wmma::load_matrix_sync(b_frag[i][j], shared_real + i * 128 * 16 + j * 16, 128);
            }
        }

        __syncthreads();

        #pragma unroll
            for (int j = 0; j < 8; j++)
            {
                wmma::fill_fragment(acc_frag_real[j], 0.0f);

                for (int k = 0; k < K; k++)
                {
                    wmma::mma_sync(acc_frag_real[j], a_frag_real[k], b_frag[k][j], acc_frag_real[j]);
                }
            }

    #pragma unroll

            for (int j = 0; j < 8; j++)
            {
                wmma::fill_fragment(acc_frag_imag[j], 0.0f);

                for (int k = 0; k < K; k++)
                {
                    wmma::mma_sync(acc_frag_imag[j], a_frag_imag[k], b_frag[k][j], acc_frag_imag[j]);
                }
            }

            float2 tmp_real, tmp_imag;
    #pragma unroll
            for (int j = 0; j < 8; j++)
            {
                for (int k = 0; k < acc_frag_real[j].num_elements / 2; k++)
                {
                    tmp_real = reinterpret_cast<float2 *>(acc_frag_real[j].x)[k];
                    tmp_imag = reinterpret_cast<float2 *>(acc_frag_imag[j].x)[k];
                
                    reinterpret_cast<float2 *>(acc_frag_real[j].x)[k] = tmp_real * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[j].x)[k]) - tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[j].x)[k]);
                    reinterpret_cast<float2 *>(acc_frag_imag[j].x)[k] = tmp_real * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag[j].x)[k]) + tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real[j].x)[k]);
                }
            }

            for (int j = 0; j < 8; j++)
            {
                wmma::store_matrix_sync(reinterpret_cast<float*>(shared_real) + threadIdx.y * 128 * 16 + j * 16, acc_frag_real[j], 128, wmma::mem_row_major);
            }

            __syncthreads();

    #pragma unroll
            for (int i = threadIdx.y; i < N; i+=blockDim.y)
            {
                for(int j=0; j< 2; j++){
                    idx = i * 32 * 2 * gridDim.x + j * blockDim.x + blockIdx.x * 64 + threadIdx.x;
                    shared_offset = i * 64 + threadIdx.x + j * blockDim.x; 
                    out_real[idx + out_offset + out_t_offset] = __float22bfloat162_rn(reinterpret_cast<float2*>(shared_real)[shared_offset]);
                }
            }

            __syncthreads();


            for (int j = 0; j < 8; j++)
            {
                wmma::store_matrix_sync(reinterpret_cast<float*>(shared_real) + threadIdx.y * 128 * 16 + j * 16, acc_frag_imag[j], 128, wmma::mem_row_major);
            }

            __syncthreads();

    #pragma unroll
            for (int i = threadIdx.y; i < N; i+=blockDim.y)
            {
                for(int j=0; j< 2; j++){
                    idx = i * 32 * 2 * gridDim.x + j * blockDim.x + blockIdx.x * 64 + threadIdx.x;
                    shared_offset = i * 64 + threadIdx.x + j * blockDim.x; 
                    out_imag[idx + out_offset + out_t_offset] = __float22bfloat162_rn(reinterpret_cast<float2*>(shared_real)[shared_offset]);
                }
            }
    }
}

template<int K>
__global__ void butterfly_cuda_kernel_16(
    const __hip_bfloat162 *__restrict__ x,
    const __hip_bfloat162 *__restrict__ x_gate,
    const __hip_bfloat16 *__restrict__ d_f_real,
    const __hip_bfloat16 *__restrict__ d_f_imag,
    const __hip_bfloat162 *__restrict__ twiddle_factors_real,
    const __hip_bfloat162 *__restrict__ twiddle_factors_imag,
    __hip_bfloat162 *__restrict__ out_real,
    __hip_bfloat162 *__restrict__ out_imag,
    uint B,
    uint H,
    int M)
{
    const int max_idx = M / 2; //actually should be -1 since indices are 0-based but we are using < instead of <=
    const int N  = 16;
    const int offset  =  blockIdx.y * H * M / 2 + blockIdx.z * M / 2; 
    const int out_offset = blockIdx.y * H * N * blockDim.x * gridDim.x + blockIdx.z * N * blockDim.x * gridDim.x;

    

    __shared__ __hip_bfloat16 x_shared[N * 64];
    __shared__ __hip_bfloat16 d_f_real_shared[N * N];
    __shared__ __hip_bfloat16 d_f_imag_shared[N * N];
    __shared__ __hip_bfloat16 twiddles_real_shared[N * 64];
    __shared__ __hip_bfloat16 twiddles_imag_shared[N * 64];
    __shared__ float out_real_shared[N * 64];
    __shared__ float out_imag_shared[N * 64];

    // #pragma unroll
    for (int i = threadIdx.y; i < N; i++)
    {
        int idx = i * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
        int shared_offset = i * blockDim.x + threadIdx.x;

        if(x_gate != nullptr){
            reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? __hmul2(x[idx + offset], x_gate[idx + offset]) : __floats2bfloat162_rn(0.0f, 0.0f);
        }else{
            reinterpret_cast<__hip_bfloat162 *>(x_shared)[shared_offset] = idx < max_idx ? x[idx + offset] : __floats2bfloat162_rn(0.0f, 0.0f);
        }
        reinterpret_cast<__hip_bfloat162 *>(twiddles_real_shared)[shared_offset] = twiddle_factors_real[idx];
        reinterpret_cast<__hip_bfloat162 *>(twiddles_imag_shared)[shared_offset] = twiddle_factors_imag[idx];

        // #pragma unroll
        if(threadIdx.x  < 16 ){
            shared_offset = i * 16 + threadIdx.x;
            d_f_real_shared[shared_offset] = d_f_real[shared_offset];
            d_f_imag_shared[shared_offset] = d_f_imag[shared_offset];
        }
    }

    __syncthreads();

    if (threadIdx.y < 4)
    {
        float2 tmp_real, tmp_imag;

        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_real;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_real;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::row_major> tw_frag_imag;
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> a_frag_imag;
        wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> b_frag;
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_real;
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag_imag;


        wmma::load_matrix_sync(a_frag_real, d_f_real_shared, N);
        wmma::load_matrix_sync(a_frag_imag, d_f_imag_shared, N);
        wmma::load_matrix_sync(b_frag, x_shared + threadIdx.y * 16, 64);
        wmma::load_matrix_sync(tw_frag_real, twiddles_real_shared + threadIdx.y * 16, 64);
        wmma::load_matrix_sync(tw_frag_imag, twiddles_imag_shared + threadIdx.y * 16, 64);
 


        wmma::fill_fragment(acc_frag_real, 0.0f);


        wmma::mma_sync(acc_frag_real, a_frag_real, b_frag, acc_frag_real);



        wmma::fill_fragment(acc_frag_imag, 0.0f);


         wmma::mma_sync(acc_frag_imag, a_frag_imag, b_frag, acc_frag_imag);
    

#pragma unroll
        for (int k = 0; k < acc_frag_real.num_elements / 2; k++)
        {
            tmp_real = 	reinterpret_cast<float2 *>(acc_frag_real.x)[k];
            tmp_imag = 	reinterpret_cast<float2 *>(acc_frag_imag.x)[k];
            reinterpret_cast<float2 *>(acc_frag_real.x)[k] = 	tmp_real  * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real.x)[k]) - tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag.x)[k]);
            reinterpret_cast<float2 *>(acc_frag_imag.x)[k] =  tmp_real  * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_imag.x)[k]) + tmp_imag * __bfloat1622float2(reinterpret_cast<__hip_bfloat162 *>(tw_frag_real.x)[k]);
        }
        wmma::store_matrix_sync(out_real_shared + threadIdx.y * 16, acc_frag_real, 64, wmma::mem_row_major);
        wmma::store_matrix_sync(out_imag_shared + threadIdx.y * 16, acc_frag_imag, 64, wmma::mem_row_major);

    }
    __syncthreads();

#pragma unroll
    for (int i = threadIdx.y; i < N; i++)
    {
        int idx = i * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;;
        out_real[out_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_real_shared)[i * 32 + threadIdx.x]);
        out_imag[out_offset + idx] = __float22bfloat162_rn(reinterpret_cast<float2*>(out_imag_shared)[i * 32 + threadIdx.x]);
    }
}

std::vector<torch::Tensor> butterfly_padded_bf16_cuda(
    torch::Tensor x,
    torch::Tensor d_f_real,
    torch::Tensor d_f_imag,
    torch::Tensor twiddle_factors_real,
    torch::Tensor twiddle_factors_imag,
    int M,
    std::optional<at::Tensor> x_gate = std::nullopt
    )
{

    uint B = x.size(0);
    uint H = x.size(1);

    uint d_f_size = d_f_real.size(1);

    uint N = x.size(2);

    //need to make sure that N is less that the M to which we are padding 
    assert(N <= d_f_size * M);

    dim3 gridDim;
    dim3 blockDim;

    gridDim.y = B;
    gridDim.z = H;

    blockDim.x = 32;
    blockDim.y = 4;

    torch::Tensor out_real = torch::empty({B, H, d_f_size * M}, x.options());
    torch::Tensor out_imag = torch::empty({B, H, d_f_size * M}, x.options());

    gridDim.x = 512 / (32 * 1024/ M);

    const int K = ceil(N / (1.0 * 16 * M));

    switch (d_f_size)
    {
        case 16:
            butterfly_cuda_kernel_16<1><<<gridDim, blockDim>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat16 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat16 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
        case 32:
            switch(K){
                case 1:
                    butterfly_cuda_kernel_32<1><<<gridDim, blockDim>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat16 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat16 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
                case 2:
                    butterfly_cuda_kernel_32<2><<<gridDim, blockDim>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat16 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat16 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                default:
                    printf("Invalid K, df size 32: %d\n", K);
            }
            break;
        case 64:
            gridDim.z = H / 16;

            switch(K){
                case 1:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_64<1>), hipFuncAttributeMaxDynamicSharedMemorySize, 78000);
                    butterfly_cuda_kernel_64<1><<<gridDim, blockDim, 78000>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
                case 2:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_64<2>), hipFuncAttributeMaxDynamicSharedMemorySize, 78000);
                    butterfly_cuda_kernel_64<2><<<gridDim, blockDim, 78000>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
                case 3:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_64<3>), hipFuncAttributeMaxDynamicSharedMemorySize, 78000);
                    butterfly_cuda_kernel_64<3><<<gridDim, blockDim, 78000>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
                case 4:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_64<4>), hipFuncAttributeMaxDynamicSharedMemorySize, 78000);
                    butterfly_cuda_kernel_64<4><<<gridDim, blockDim, 78000>>>(
                    static_cast<__hip_bfloat162 *>(x.data_ptr()),
                    x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                    static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                    static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                    B,
                    H,
                    N);
                    break;
                default:
                    printf("Invalid K, df size 64: %d\n", K);
            }   
            break;
        case 128:
            blockDim.x = 32;
            blockDim.y = 8;
            gridDim.x = 256 / (32 * 1024/ M);
            gridDim.z = H / 16;
            switch(K){
                case 1:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<1>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
                    butterfly_cuda_kernel_128<1><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                        break;
                case 2:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<2>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
                    butterfly_cuda_kernel_128<2><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 3:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<3>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<3><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 4:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<4>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<4><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 5:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<5>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<5><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 6:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<6>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<6><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 7:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<7>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<7><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                case 8:
                    hipFuncSetAttribute(reinterpret_cast<const void*>(&butterfly_cuda_kernel_128<8>), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

                    butterfly_cuda_kernel_128<8><<<gridDim, blockDim, 65536>>>(
                        static_cast<__hip_bfloat162 *>(x.data_ptr()),
                        x_gate ? static_cast<__hip_bfloat162 *>(x_gate.value().data_ptr()) : nullptr,
                        static_cast<__hip_bfloat162 *>(d_f_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(d_f_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(twiddle_factors_imag.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_real.data_ptr()),
                        static_cast<__hip_bfloat162 *>(out_imag.data_ptr()),
                        B,
                        H,
                        N);
                    break;
                default:
                    printf("Invalid K, df size 128: %d\n", K);

            }
            break;

        default:
        printf("Not yet implemented \n");
            break;
    }

    return {out_real, out_imag};
}